#include "hip/hip_runtime.h"
// ### Adrian's
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, , p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

//ex1
cv::Mat kernel(float sigma){
    int r = ceil(3*sigma);
    float sigma2=powf(sigma,2);

    cv::Mat kernel(2*r+1,2*r+1,CV_32FC1);

    for (int i = 0; i <= r; ++i)
    {
        for (int j = 0; j <= r; ++j)
        {
            float value=1/(2*M_PI*sigma2) * expf( -( powf(i,2)+powf(j,2) ) / (2*sigma2) );
            kernel.at<float>(r+i,r+j)=value;
            kernel.at<float>(r-i,r+j)=value;
            kernel.at<float>(r+i,r-j)=value;
            kernel.at<float>(r-i,r-j)=value;
        }
    }

    float s = sum(kernel)[0];
    kernel/=s;

    //std::cout<<"kernel:"<<std::endl;
    //std::cout<<kernel<<std::endl;

    return kernel;
}

//ex2
void imagesc(std::string name, cv::Mat mat){
    double min,max;
    cv::minMaxLoc(mat,&min,&max);
    cv::Mat  kernel_prime = mat/max;
    showImage(name, kernel_prime, 50,50);
}

//ex3
cv::Mat convolution(cv::Mat k, cv::Mat u){
  // width and height image
    int w=u.cols;
    int h=u.rows;

    // width and height kernel
    int wk=k.cols;
    int hk=k.rows;

    int rx=wk/2;
    int ry=hk/2;

    cv::Mat out(h,w,u.type());

    // loop over all pixels
    for (int x = 0; x < w; ++x)
      {
	for (int y = 0; y < h; ++y)
	  {
	    float val=0;

	    // do convolution for every pixel
	    for (int i = 0; i < wk; ++i)
	      {
		for (int j = 0; j < hk; ++j)
		  {
		    int y_index = y-j+ry;
		    int x_index = x-i+rx;

		    // check indices - do clamping if necessary
		    if (y_index < 0)
		      y_index = 0;
		    else if(y_index >= h)
		      y_index = h-1;

		    if (x_index < 0)
		      x_index = 0;
		    else if (x_index >= w)
		      x_index = w-1;

		    val+=k.at<float>(j,i)*u.at<float>(y_index,x_index);			    		
		  }
	      }
	    out.at<float>(y,x)=val;
	  }
      }
    return out;
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    




    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << " nc="<<nc <<endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOut2(h,w,mIn.type()); 
    cv::Mat mOut3(h,w,mIn.type()); 

    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    cv::Mat mOut4(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed





    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[n];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[n];
    float *imgOut2 = new float[n];
    float *imgDivergence = new float[n];

    size_t n_OneChannel = (size_t)w*h*1;
    float *imgLaplacian = new float[n_OneChannel]; //only one channel






    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    float sigma = 4.0f;
    cv::Mat k=kernel(sigma);
    
    imagesc("Kernel", k);


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    std::cout<<"after showing input image"<<std::cout;


    cv::Mat blurred=convolution(k,mIn);

    // show output image: first convert to interleaved opencv format from the layered raw array
    showImage("Blurred", blurred, 100+w+40, 100);

    std::cout<<"after showing blurred image"<<std::cout;

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    /*convert_mat_to_layered (imgIn, mIn);

	//GPU:
	
	float *d_imgIn, *d_v2, *d_v1, *d_divergence, *d_laplacian;

	hipMalloc(&d_imgIn, n * sizeof(float) );CUDA_CHECK;
	hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    hipMalloc(&d_v1, n * sizeof(float) ); CUDA_CHECK;
	hipMalloc(&d_v2, n * sizeof(float) ); CUDA_CHECK;
    hipMalloc(&d_divergence, n * sizeof(float) ); CUDA_CHECK;

    hipMalloc(&d_laplacian, n_OneChannel * sizeof(float)); CUDA_CHECK; //notice: only one channel


	dim3 block = dim3(32,8,1);
	dim3 grid = dim3((w + block.x - 1 ) / block.x,(h + block.y - 1 ) / block.y, 1);

    cout <<"grids: "<< grid.x<< "x" <<grid.y<<endl;
	
    gradient <<<grid,block>>> (d_imgIn, d_v1, d_v2, w, h, nc);CUDA_CHECK;
	hipDeviceSynchronize();CUDA_CHECK;
    divergence<<<grid,block>>> (d_v1, d_v2, d_divergence, w, h, nc);CUDA_CHECK;
    hipDeviceSynchronize();CUDA_CHECK;
    l2norm<<<grid,block>>> (d_divergence, d_laplacian, w, h, nc);CUDA_CHECK;
    hipDeviceSynchronize();CUDA_CHECK;


	hipMemcpy(imgOut, d_v1, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
    hipMemcpy(imgOut2, d_v2, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
    hipMemcpy(imgDivergence, d_divergence, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
    hipMemcpy(imgLaplacian, d_laplacian, n_OneChannel * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;

	hipFree(d_v1);CUDA_CHECK;
    hipFree(d_v2);CUDA_CHECK;
    hipFree(d_divergence);CUDA_CHECK;
	hipFree(d_imgIn);CUDA_CHECK;
    hipFree(d_laplacian);CUDA_CHECK;
*/




/*
    convert_layered_to_mat(mOut2, imgOut2);
    showImage("Gradient_Y", mOut2, 100+2*w+40, 100);

    convert_layered_to_mat(mOut3, imgDivergence);
    showImage("Divergence", mOut3, 100+3*w+40, 100);

    convert_layered_to_mat(mOut4, imgLaplacian);
    showImage("Laplacian", mOut4, 100+4*w+40, 100);
    */

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif



/*
    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    delete[] imgDivergence;
    delete[] imgOut2;
    */

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



