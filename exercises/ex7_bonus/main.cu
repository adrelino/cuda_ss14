// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###
// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "aux.h"
#include "hipblas.h"

string get_cublas_error(hipblasStatus_t stat) {
  switch(stat)
    {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }

  return "Unknown error";
}

void cublas_check(hipblasStatus_t stat) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    cerr << "Received error: " << get_cublas_error(stat) << endl;
  }
}

int main(int argc, char **argv)
{
  hipDeviceSynchronize(); CUDA_CHECK;
  
  // alloc and init input arrays on host (CPU)
  int n = 20;
    
  float *h_a = new float[n];
  float *h_b = new float[n];
  float *h_c = new float[n];

  for(int i=0; i<n; i++)
    {
      h_a[i] = i;
      h_b[i] = (i%5)+1;
      h_c[i] = 0;
    }
    
  // CPU computation
  for(int i=0; i<n; i++) h_c[i] = h_a[i] + h_b[i];

  // print result
  cout << "CPU:"<<endl;
  for(int i=0; i<n; i++) cout << i << ": " << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << endl;
  cout << endl;
  // init c
  for(int i=0; i<n; i++) h_c[i] = 0;
    
  // GPU computation
  // allocate memory on GPU
  size_t size_arr = n * sizeof(float);
  float *d_a, *d_b;

  hipMalloc(&d_a, size_arr);
  CUDA_CHECK;

  hipMalloc(&d_b, size_arr);
  CUDA_CHECK;
    
  // CUBLAS stuff
  hipblasStatus_t stat;
  hipblasHandle_t handle;    
  // get CUBLAS context
  stat = hipblasCreate(&handle);
  cublas_check(stat);
    
  // copy the stuff from h_a, h_b to d_a, d_b
  stat = hipblasSetVector(n, sizeof(*h_a), h_a, 1, d_a, 1);
  cublas_check(stat);
    
  stat = hipblasSetVector(n, sizeof(*h_b), h_b, 1, d_b, 1);
  cublas_check(stat);
    
  // run the cublas algorithm
  float alph = 1.0f;
  stat = hipblasSaxpy(handle, n, &alph, d_a, 1, d_b, 1);
  cublas_check(stat);    

  // get vector back from CUBLAS
  stat = hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);
  cublas_check(stat);    
    
  // cuda cleanup
  hipFree(d_a);
  CUDA_CHECK;

  hipFree(d_b);
  CUDA_CHECK;

  // cublas cleanup
  hipblasDestroy(handle);
    
  // print result
  cout << "GPU using CUBLAS library:"<<endl;
  for(int i=0; i<n; i++) cout << i << ": " << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << endl;
  cout << endl;

  // free CPU arrays
  delete[] h_a;
  delete[] h_b;
  delete[] h_c;
}



