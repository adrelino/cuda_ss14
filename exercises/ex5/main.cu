#include "hip/hip_runtime.h"
// ### Adrian's
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

//                       in             out      out
__device__ void gradient(float *imgIn, float *v1, float *v2, int w, int h, int nc){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    if(x>w || y>h) return;

    int xPlus = x + 1;
    if(xPlus>=w) xPlus=w-1;

    int yPlus = y + 1;
    if(yPlus>=h) yPlus=h-1;

    for (int i = 0; i < nc; ++i)
    {
        v1[x+ y*w +i*w*h]=imgIn[xPlus+ y*w + i*w*h]-imgIn[x+ y*w + i*w*h];
        v2[x+ y*w +i*w*h]=imgIn[x+ yPlus*w + i*w*h]-imgIn[x+ y*w + i*w*h];

    }
}

//                         in        in         out
__device__ void divergence(float *v1, float *v2, float *imgOut, int w, int h, int nc){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    if(x>w || y>h) return;

    int xMinus = x - 1;
    if(xMinus<0) xMinus=0;

    int yMinus = y - 1;
    if(yMinus<0) yMinus=0;

    for (int i = 0; i < nc; ++i)
    {
        float backv1_x=v1[x+ y*w +i*w*h]-v1[xMinus+ y*w + i*w*h];
        float backv2_y=v2[x+ y*w + i*w*h]-v2[x+ yMinus*w + i*w*h];
        imgOut[x+ y*w +i*w*h]=backv1_x+backv2_y;
    }
}

//                     in           out
__device__ void l2norm(float *imgIn, float *imgOut, int w, int h, int nc){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    if(x>w || y>h) return;

    float c=0;

    for (int i = 0; i < nc; ++i)
    {
        c+=powf(imgIn[x+ y*w +i*w*h],2);
    }

    c=sqrtf(c);

    imgOut[x+ y*w]=c; //channel is 0 -> grayscale
}

__global__ void gpuEntry(float* d_imgIn, float* d_v1, float* d_v2, float* d_divergence, float* d_laplacian, int w, int h, int nc){
    gradient (d_imgIn, d_v1, d_v2, w, h, nc);
    divergence (d_v1, d_v2, d_divergence, w, h, nc);
    l2norm (d_divergence, d_laplacian, w, h, nc);
}

float GetAverage(float dArray[], int iSize) {
    float dSum = dArray[0];
    for (int i = 1; i < iSize; ++i) {
        dSum += dArray[i];
    }
    return dSum/iSize;
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    




    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << " nc="<<nc <<endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOut2(h,w,mIn.type()); 
    cv::Mat mOut3(h,w,mIn.type()); 

    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    cv::Mat mOut4(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[n];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[n];
    float *imgOut2 = new float[n];
    float *imgDivergence = new float[n];

    size_t n_OneChannel = (size_t)w*h*1;
    float *imgLaplacian = new float[n_OneChannel]; //only one channel






    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    float *tg, *tg2;
    tg=(float*)malloc(repeats*sizeof(float));
    tg2=(float*)malloc(repeats*sizeof(float));

	//GPU:
	for(int i=0;i<repeats;i++){
        Timer timergpu, timergpu2; 
        timergpu.start();

    	float *d_imgIn, *d_v2, *d_v1, *d_divergence, *d_laplacian;

    	hipMalloc(&d_imgIn, n * sizeof(float) );CUDA_CHECK;
    	hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

        hipMalloc(&d_v1, n * sizeof(float) ); CUDA_CHECK;
    	hipMalloc(&d_v2, n * sizeof(float) ); CUDA_CHECK;
        hipMalloc(&d_divergence, n * sizeof(float) ); CUDA_CHECK;

        hipMalloc(&d_laplacian, n_OneChannel * sizeof(float)); CUDA_CHECK; //notice: only one channel

        timergpu2.start();

    	dim3 block = dim3(32,8,1);
    	dim3 grid = dim3((w + block.x - 1 ) / block.x,(h + block.y - 1 ) / block.y, 1);

    	
        gpuEntry<<<grid,block>>> (d_imgIn, d_v1, d_v2, d_divergence, d_laplacian, w, h, nc);CUDA_CHECK;
        hipDeviceSynchronize();CUDA_CHECK;
        
        timergpu2.end();
        tg2[i] = timergpu2.get();

    	hipMemcpy(imgOut, d_v1, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
        hipMemcpy(imgOut2, d_v2, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
        hipMemcpy(imgDivergence, d_divergence, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
        hipMemcpy(imgLaplacian, d_laplacian, n_OneChannel * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;

    	hipFree(d_v1);CUDA_CHECK;
        hipFree(d_v2);CUDA_CHECK;
        hipFree(d_divergence);CUDA_CHECK;
    	hipFree(d_imgIn);CUDA_CHECK;
        hipFree(d_laplacian);CUDA_CHECK;

        timergpu.end(); 
        tg[i] = timergpu.get();

    }

    cout << "avg time gpu: " << GetAverage(tg, repeats)*1000 << " ms" << endl;
    cout << "avg time gpu allocfree: " << GetAverage(tg2, repeats)*1000 << " ms" << endl;

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Gradient_X", mOut, 100+w+40, 100);

    convert_layered_to_mat(mOut2, imgOut2);
    showImage("Gradient_Y", mOut2, 100+2*w+40, 100);

    convert_layered_to_mat(mOut3, imgDivergence);
    showImage("Divergence", mOut3, 100+3*w+40, 100);

    convert_layered_to_mat(mOut4, imgLaplacian);
    showImage("Laplacian", mOut4, 100+4*w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    delete[] imgDivergence;
    delete[] imgOut2;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



