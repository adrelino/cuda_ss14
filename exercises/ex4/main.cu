#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070


#include "helper.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA

__device__ void img_thresholding(float *d_imgIn, float *d_imgOut, size_t w, size_t h, size_t nc, float thresh) {
  
  // get the pixel id
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  
  size_t nr_pixels = w * h;

  if (x > w-1 || y > h-1)
    return;

  int pxid = x + y*w;
  float val = 0.0f;
    
  for(int i=0; i < nc; ++i)
    val += d_imgIn[pxid + i * nr_pixels];

  val /= nc;
    
  if (val >= thresh) {
    // red channel
    d_imgOut[pxid + (0 * nr_pixels)] = 1.0f;
    // green channel
    d_imgOut[pxid + (1 * nr_pixels)] = 0.0f;
    // blue channel
    d_imgOut[pxid + (2 * nr_pixels)] = 0.0f;    
  }
  else {
    // red channel
    d_imgOut[pxid + (0 * nr_pixels)] = 0.0f;
    // green channel
    d_imgOut[pxid + (1 * nr_pixels)] = 0.3f;
    // blue channel
    d_imgOut[pxid + (2 * nr_pixels)] = 0.7f;        
  }
}

__global__ void kernel_call(float *d_imgIn, float *d_imgOut, size_t w, size_t h, size_t nc, float thresh) {
  img_thresholding(d_imgIn, d_imgOut, w, h, nc, thresh);
}

__host__ float calc_average_time(float *arr, int n) {
  float cum_sum = 0.0f;
  for (int i = 0; i < n; ++i)
    cum_sum += arr[i];

  return cum_sum / n;
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;


    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed
    // get the threshold parameter from the command line
    float thresh = 0.13f;
    getParam("thresh", thresh, argc, argv);

    // Init camera / Load input image
#ifdef CAMERA
    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    size_t w = static_cast<size_t>(mIn.cols);         // width
    size_t h = static_cast<size_t>(mIn.rows);         // height
    size_t nc = static_cast<size_t>(mIn.channels());  // number of channels
    cout << "image: " << w << " x: " << h << endl;
    
    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed


    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t size_input_image = (size_t)w*h*nc;
    float *imgIn  = new float[size_input_image];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    size_t size_output_image = (size_t)w*h*mOut.channels();
    float *imgOut = new float[size_output_image];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    float *time_with_memory_calls = new float[repeats];
    float *time_gpu_only = new float[repeats];    
    
    for (int i=0; i < repeats; ++i) {
      Timer timer_memory_calls, timer_gpu_only;
      timer_memory_calls.start();
      
      // ###
      // ###
      // ### Main computation
      // ###
      // ###
      // create array for cuda
      float *d_imgIn, *d_imgOut;

      size_t nr_bytes_input_image = size_input_image * sizeof(float);
      hipMalloc(&d_imgIn, nr_bytes_input_image);
      CUDA_CHECK;

      size_t nr_bytes_output_image = size_output_image * sizeof(float);
      hipMalloc(&d_imgOut, nr_bytes_output_image);
      CUDA_CHECK;

      hipMemset(d_imgOut, 0, nr_bytes_output_image);
      CUDA_CHECK;

      // copy layered images to device
      hipMemcpy(d_imgIn, imgIn, nr_bytes_input_image, hipMemcpyHostToDevice);
      CUDA_CHECK;

      timer_gpu_only.start();

      // compute the appropiate dimensions for the grid/block
      dim3 block_size = dim3(32, 4, 1);
      dim3 grid_size = dim3((w + block_size.x - 1) / block_size.x, (h + block_size.y - 1) / block_size.y, 1);
    
      kernel_call<<<grid_size, block_size>>>(d_imgIn, d_imgOut, w, h, nc, thresh);
      CUDA_CHECK;
      hipDeviceSynchronize();

      timer_gpu_only.end();
      // save time in seconds
      time_gpu_only[i] = timer_gpu_only.get();
      
      // copy back from cuda memory
      hipMemcpy(imgOut, d_imgOut, nr_bytes_output_image, hipMemcpyDeviceToHost);
      CUDA_CHECK;

      // hipFree
      hipFree(d_imgIn);
      CUDA_CHECK;

      hipFree(d_imgOut);
      CUDA_CHECK;

      timer_memory_calls.end();
      time_with_memory_calls[i] = timer_memory_calls.get();  // elapsed time in seconds
    }

    cout << "avg time gpu: " << calc_average_time(time_with_memory_calls, repeats)*1000 << " ms" << endl;
    cout << "avg time gpu alloc free: " << calc_average_time(time_gpu_only, repeats)*1000 << " ms" << endl;

    delete[] time_with_memory_calls;
    delete[] time_gpu_only;

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



