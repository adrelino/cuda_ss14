#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ###
// ###
// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

cv::Mat kernel(float sigma){
    int r = ceil(3*sigma);
    float sigma2=powf(sigma,2);

    cv::Mat kernel(2*r+1,2*r+1,CV_32FC1);

    for (int i = 0; i <= r; ++i)
    {
        for (int j = 0; j <= r; ++j)
        {
            float value=1/(2*M_PI*sigma2) * expf( -( powf(i,2)+powf(j,2) ) / (2*sigma2) );
            kernel.at<float>(r+i,r+j)=value;
            kernel.at<float>(r-i,r+j)=value;
            kernel.at<float>(r+i,r-j)=value;
            kernel.at<float>(r-i,r-j)=value;
        }
    }

    float s = sum(kernel)[0];
    kernel/=s;

    return kernel;
}

// TODO: move it back to __device__
__global__ void convolutionGPU(float *imgIn, float *GK, float *imgOut, int w, int h, int nc, int wk, int hk){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;
    size_t k = wk;//==hk

    int rx=wk/2;
    int ry=hk/2;


    if(x>=w || y>=h) return; //check for blocks

    for(unsigned int c=0;c<nc;c++) {
        float sum=0;
        for(unsigned int i=0;i<k;i++){
            unsigned int x_new;
            if(x+rx<i) x_new=rx;
            else if(x+rx-i>=w) x_new=w+rx-1;
            else x_new=x+rx-i;
            for(unsigned int j=0;j<k;j++){
                unsigned int y_new;
                if(y+ry<j) y_new=0;
                else if(y+ry-j>=h) y_new=h+ry-1;
                else y_new=y+ry-j;
                sum+=GK[i+j*k]*imgIn[x_new+y_new*w+w*h*c];
                // if(sum<0) cout << "fuck" << endl;
            }
        }
        imgOut[x+w*y+w*h*c]=sum;
    }
}

__global__ void computeSpatialDerivatives(float *d_img, float *d_dx, float *d_dy, int w, int h, int nc) {

  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;

  // if outside of image --> return
  if (x > w || y > h)
    return;

  // calc indices
  int xPlus1 = x+1;
  int xMinus1 = x-1;

  int yPlus1 = y+1;
  int yMinus1 = y-1;

  // do clamping
  if (xPlus1 >= w)
    xPlus1 = w-1;
  if (yPlus1 >= h)
    yPlus1 = h-1;

  if (xMinus1 < 0)
    xMinus1 = 0;
  if (yMinus1 < 0)
    yMinus1 = 0;

  // calc derivatives
  for (int c = 0; c < nc; ++c) {
    // x-derivatives
    d_dx[x + y*w + c*w*h] = (3*d_img[xPlus1 + yPlus1*w + c*w*h] +
                             10*d_img[xPlus1 + y*w + c*w*h] +
                             3*d_img[xPlus1 + yMinus1*w + c*w*h] -
                             3*d_img[xMinus1 + yPlus1*w + c*w*h] -
                             10*d_img[xMinus1 + y*w + c*w*h] -
                             3*d_img[xMinus1 + yMinus1*w + c*w*h]) / 32.0f;

    // y-derivatives
    d_dy[x + y*w + c*w*h] = (3*d_img[xPlus1 + yPlus1*w + c*w*h] +
                             10*d_img[x + yPlus1*w + c*w*h] +
                             3*d_img[xMinus1 + yPlus1*w + c*w*h] -
                             3*d_img[xPlus1 + yMinus1*w + c*w*h] -
                             10*d_img[x + yMinus1*w + c*w*h] -
                             3*d_img[xMinus1 + yMinus1*w + c*w*h]) / 32.0f;

  }
}

__global__ void createStructureTensor(float *d_dx, float *d_dy, int w, int h, int nc, float *d_m11, float *d_m12, float *d_m22) {
  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x > w || y > h)
    return;

  for(int c = 0; c < nc; ++c) {
    // caution: only possible if arrays were memsetted to zero!
    d_m11[x + y * w] += d_dx[x + y * w + c*w*h] * d_dx[x + y * w + c*w*h];
    d_m12[x + y * w] += d_dx[x + y * w + c*w*h] * d_dy[x + y * w + c*w*h];
    d_m22[x + y * w] += d_dy[x + y * w + c*w*h] * d_dy[x + y * w + c*w*h];
  }
}

__device__ void compute_eigenvalues2x2(float a, float b, float c, float d, float *lambda1, float *lambda2) {
  float trace = a + d;
  float determinant = a*d - b*c;

  *lambda1 = trace / 2.0f + sqrtf((trace*trace)/(4.0f-determinant));
  *lambda2 = trace / 2.0f - sqrtf((trace*trace)/(4.0f-determinant));
}

// TODO: use alph and beta as constant variables in CUDA
__global__ void feature_detection(float *d_imgIn, float *d_imgOut, int w, int h, float *d_m11, float *d_m12, float *d_m22, float alph, float beta) {

  size_t x = threadIdx.x + blockDim.x * blockIdx.x;
  size_t y = threadIdx.y + blockDim.y * blockIdx.y;

  if (x > w || y > h)
    return;

  float lambda1, lambda2;

  compute_eigenvalues2x2(d_m11[x + y*w], d_m12[x + y*w], d_m12[x + y*w], d_m22[x + y*w], &lambda1, &lambda2);

  if (lambda1 >= alph) {
    d_imgOut[x + y * w] = 1.0f;
    d_imgOut[x + y * w + 1*w*h] = 0.0f;
    d_imgOut[x + y * w + 2*w*h] = 0.0f;
  }
  else if ((lambda2 >= alph) && (lambda1 <= beta) && (alph > beta)) {
    d_imgOut[x + y * w] = 0.0f;
    d_imgOut[x + y * w + 1*w*h] = 1.0f;
    d_imgOut[x + y * w + 2*w*h] = 0.0f;
  }
  else {
    // TODO: make this more general and dependent of nc
    d_imgOut[x + y * w] = d_imgIn[x + y *w] * 0.5f;
    d_imgOut[x + y * w + 1*w*h] = d_imgIn[x + y *w + 1*w*h] * 0.5f;
    d_imgOut[x + y * w + 2*w*h] = d_imgIn[x + y *w + 2*w*h] * 0.5f;
  }
}

__global__ void calcStructureTensor(float *d_imgIn, float *GK, int w, int h, int nc, float *d_m11, float *d_m12, float *d_m22) {

}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif

    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;

    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed
    float sigma = 0.5f;
    getParam("sigma", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;

    float alph = 10E-2;
    getParam("alpha", alph, argc, argv);
    cout << "alpha: " << alph << endl;

    float beta = 10E-3;
    getParam("beta", beta, argc, argv);
    cout << "beta: " << beta << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
        cv::VideoCapture camera(0);
        if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
        camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
        camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;

#else

    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    // smoothed image and directional derivatives
    cv::Mat mSmooth(h,w,mIn.type());
    cv::Mat mImgV1(h, w, mIn.type());
    cv::Mat mImgV2(h, w, mIn.type());

    // components of M (grayscale images)
    cv::Mat mImgM11(h, w, CV_32FC1);
    cv::Mat mImgM12(h, w, CV_32FC1);
    cv::Mat mImgM22(h, w, CV_32FC1);

    // feature map
    cv::Mat mImgFeatureMap(h, w, CV_32FC3);

    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // smoothed image
    float *imgSmooth = new float[(size_t)w*h*mSmooth.channels()];

    // derivatives in x-direction
    float *imgV1 = new float[(size_t)w*h*mImgV1.channels()];

    // derivatives in y-direction
    float *imgV2 = new float[(size_t)w*h*mImgV2.channels()];

    float *imgM11 = new float[(size_t)w*h];
    float *imgM12 = new float[(size_t)w*h];
    float *imgM22 = new float[(size_t)w*h];

    float *imgFeatureMap = new float[(size_t)w*h*3];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    // create kernel for smoothing
    cv::Mat k = kernel(sigma);
    int wk = k.cols;
    int hk = k.rows;
    size_t nr_pixels_kernel = wk * hk;

    float *imgKernel = new float[nr_pixels_kernel];
    convert_mat_to_layered(imgKernel, k);

    // GPU computation
    // transfer data to GPU
    float *d_imgIn, *d_imgKernel, *d_imgS, *d_imgV1, *d_imgV2;
    float *d_imgM11, *d_imgM12, *d_imgM22;
    float *d_imgFeatureMap;

    hipMalloc(&d_imgIn, n * sizeof(float));
    CUDA_CHECK;

    hipMalloc(&d_imgKernel, nr_pixels_kernel * sizeof(float));
    CUDA_CHECK;

    hipMalloc(&d_imgS, n * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgV1, n * sizeof(float)); CUDA_CHECK;

    hipMalloc(&d_imgV2, n * sizeof(float)); CUDA_CHECK;

    // allocate memory for structure tensor
    const size_t sz_m = w * h * sizeof(float);
    hipMalloc(&d_imgM11, sz_m); CUDA_CHECK;
    hipMalloc(&d_imgM12, sz_m); CUDA_CHECK;
    hipMalloc(&d_imgM22, sz_m); CUDA_CHECK;

    // set data structure to zero
    hipMemset(d_imgM11, 0, sz_m); CUDA_CHECK;
    hipMemset(d_imgM12, 0, sz_m); CUDA_CHECK;
    hipMemset(d_imgM22, 0, sz_m); CUDA_CHECK;

    // allocate memory for feature map
    hipMalloc(&d_imgFeatureMap, w * h * 3 * sizeof(float)); CUDA_CHECK;

    hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK;

    hipMemcpy(d_imgKernel, imgKernel, nr_pixels_kernel * sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK;

    // prepare grid
    dim3 block_size = dim3(32,4,1);
    dim3 grid_size = dim3((w + block_size.x - 1 ) / block_size.x,(h + block_size.y - 1 ) / block_size.y, 1);

    // first, smooth image using GPU
    convolutionGPU <<<grid_size, block_size>>> (d_imgIn, d_imgKernel, d_imgS, w, h, nc, wk, hk);
    CUDA_CHECK;

    hipDeviceSynchronize();
    CUDA_CHECK;

    // second, create derivatives
    computeSpatialDerivatives<<<grid_size, block_size>>>(d_imgS, d_imgV1, d_imgV2, w, h, nc);
    CUDA_CHECK;

    hipDeviceSynchronize();
    CUDA_CHECK;

    // third, create structure tensor (m11, m12, m22)
    createStructureTensor<<<grid_size, block_size>>>(d_imgV1, d_imgV2, w, h, nc, d_imgM11, d_imgM12, d_imgM22);
    CUDA_CHECK;

    hipDeviceSynchronize();
    CUDA_CHECK;

    // fourth, convolve m11, m12, m22  with our kernel
    convolutionGPU<<<grid_size, block_size>>>(d_imgM11, d_imgKernel, d_imgM11, w, h, 1, wk, hk);
    CUDA_CHECK;
    convolutionGPU<<<grid_size, block_size>>>(d_imgM12, d_imgKernel, d_imgM12, w, h, 1, wk, hk);
    CUDA_CHECK;
    convolutionGPU<<<grid_size, block_size>>>(d_imgM22, d_imgKernel, d_imgM22, w, h, 1, wk, hk);
    CUDA_CHECK;
    hipDeviceSynchronize();
    CUDA_CHECK;

    // fifth, compute feature map
    feature_detection<<<grid_size, block_size>>>(d_imgIn, d_imgFeatureMap, w, h, d_imgM11, d_imgM12, d_imgM22, alph, beta);
    CUDA_CHECK;
    hipDeviceSynchronize();
    CUDA_CHECK;

    // get smoothed image back
    hipMemcpy(imgSmooth, d_imgS, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    // get derivatives back
    hipMemcpy(imgV1, d_imgV1, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgV2, d_imgV2, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    // copy m11, m12, m22 back
    hipMemcpy(imgM11, d_imgM11, w*h*sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgM12, d_imgM12, w*h*sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;
    hipMemcpy(imgM22, d_imgM22, w*h*sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    // copy feature map
    hipMemcpy(imgFeatureMap, d_imgFeatureMap, w*h*3*sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    // free stuff
    hipFree(d_imgIn); CUDA_CHECK;
    hipFree(d_imgS); CUDA_CHECK;
    hipFree(d_imgKernel); CUDA_CHECK;

    hipFree(d_imgV1); CUDA_CHECK;
    hipFree(d_imgV2); CUDA_CHECK;

    hipFree(d_imgM11); CUDA_CHECK;
    hipFree(d_imgM12); CUDA_CHECK;
    hipFree(d_imgM22); CUDA_CHECK;

    hipFree(d_imgFeatureMap); CUDA_CHECK;

    // show input imagew
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    convert_layered_to_mat(mSmooth, imgSmooth);
    showImage("Smoothed Image", mSmooth, 100+w+40, 100);

    convert_layered_to_mat(mImgV1, imgV1);
    showImage("x-Derivative", mImgV1, 100+2*w+40, 100);

    convert_layered_to_mat(mImgV2, imgV2);
    showImage("y-Derivative", mImgV2, 100+3*w+40, 100);

    float scale_factor = 10.0f;

    convert_layered_to_mat(mImgM11, imgM11);
    mImgM11 *= scale_factor;
    showImage("m11", mImgM11, 100+4*w+40, 100);

    convert_layered_to_mat(mImgM12, imgM12);
    mImgM12 *= scale_factor;
    showImage("m12", mImgM12, 100+5*w+40, 100);

    convert_layered_to_mat(mImgM22, imgM22);
    mImgM22 *= scale_factor;
    showImage("m22", mImgM22, 100+6*w+40, 100);

    // show feature map
    convert_layered_to_mat(mImgFeatureMap, imgFeatureMap);
    showImage("Feature Map", mImgFeatureMap, 100, 300);


#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    delete[] imgKernel;

    delete[] imgSmooth;
    delete[] imgV1;
    delete[] imgV2;

    delete[] imgM11;
    delete[] imgM12;
    delete[] imgM22;

    delete[] imgFeatureMap;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
