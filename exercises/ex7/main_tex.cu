#include "hip/hip_runtime.h"
// ### Adrian's
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, markus.schlaffer@in.tum.de, p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

// create texture for storing input image
texture<float, 2, hipReadModeElementType> texRef;

cv::Mat kernel(float sigma){
    int r = ceil(3*sigma);
    float sigma2=powf(sigma,2);

    cv::Mat kernel(2*r+1,2*r+1,CV_32FC1);

    for (int i = 0; i <= r; ++i)
    {
        for (int j = 0; j <= r; ++j)
        {
            float value=1/(2*M_PI*sigma2) * expf( -( powf(i,2)+powf(j,2) ) / (2*sigma2) );
            kernel.at<float>(r+i,r+j)=value;
            kernel.at<float>(r-i,r+j)=value;
            kernel.at<float>(r+i,r-j)=value;
            kernel.at<float>(r-i,r-j)=value;
        }
    }

    float s = sum(kernel)[0];
    kernel/=s;

    return kernel;
}

__global__ void convolutionGPUTex(float *GK, float *imgOut, int w, int h, int nc, int kernel_size) {
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;
    size_t k = kernel_size;

    int rx=kernel_size/2;
    int ry=kernel_size/2;

    if(x>=w || y>=h) return; //check for blocks

    for(size_t c=0;c<nc;c++) {
        float sum=0;
        for(size_t i=0;i<k;i++){
            size_t x_new;
	    x_new=x+rx-i;
	    
            for(size_t j=0;j<k;j++){
                size_t y_new;
		y_new=y+ry-j;

		float x_tex = x_new + 0.5f;
		float y_tex = y_new + c*h;

                sum+=GK[i+j*k]*tex2D(texRef, x_tex, y_tex);
            }
        }
        imgOut[x+w*y+w*h*c]=sum;
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    float sigma=3.0f;
    getParam("sigma", sigma, argc, argv);
    if(sigma<=0) sigma=3.0f;
    cout << "sigma: " << sigma << endl;


    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << " nc="<<nc <<endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[n];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[n];

    size_t n1 = (size_t)w*h*1;
    float *imgKernel  = new float[n1];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    cv::Mat k=kernel(sigma);
    
    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered(imgIn, mIn);
    convert_mat_to_layered(imgKernel,k);

    //GPU:
    float *d_imgIn, *d_imgKernel, *d_imgOut;
    hipMalloc(&d_imgIn, n * sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    // now set up the texture stuff
    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    hipMalloc(&d_imgKernel, n1 * sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgKernel, imgKernel, n1 * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;
    
    hipMalloc(&d_imgOut, n * sizeof(float) ); CUDA_CHECK;

    // problem with gt8800
    // dim3 block = dim3(32,8,1);
    dim3 block = dim3(32,4,1);
    dim3 grid = dim3((w + block.x - 1 ) / block.x,(h + block.y - 1 ) / block.y, 1);

    hipBindTexture2D(NULL, &texRef, d_imgIn, &desc, w, nc * h, w * sizeof(d_imgIn[0]));
    CUDA_CHECK;
    
    convolutionGPUTex<<<grid,block>>>(d_imgKernel, d_imgOut, w, h, nc, k.cols);
    CUDA_CHECK;
    
    hipUnbindTexture(texRef);
    CUDA_CHECK;

    hipMemcpy(imgOut, d_imgOut, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;

    hipFree(d_imgIn);CUDA_CHECK;
    hipFree(d_imgOut);CUDA_CHECK;

    convert_layered_to_mat(mOut, imgOut);
    showImage("Convolution using Texure Memory", mOut, 100+2*w+40, 100);

    //cv::Mat blurred=convolution(k,mIn);
    // show output image: first convert to interleaved opencv format from the layered raw array
    //showImage("Blurred", blurred, 100+w+40, 100);
    //std::cout<<"after showing blurred image"<<std::cout;
    
    // ### Display your own output images here as needed
#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    //cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



