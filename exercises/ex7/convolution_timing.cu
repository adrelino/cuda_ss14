#include "hip/hip_runtime.h"
// ### Adrian's
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, , p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

typedef struct Params {
    int shw;
    int shh;
    int w;
    int h;
    int nc;
    int r;
} Params;

// create texture for storing input image
texture<float, 2, hipReadModeElementType> texRef;

//ex1
cv::Mat kernel(float sigma, int r){
    float sigma2=powf(sigma,2);

    cv::Mat kernel(2*r+1,2*r+1,CV_32FC1);

    if(r==0){
        kernel.at<float>(0,0)=1;
        return kernel;
    }

    for (int i = 0; i <= r; ++i)
    {
        for (int j = 0; j <= r; ++j)
        {
            float value=1/(2*M_PI*sigma2) * expf( -( powf(i,2)+powf(j,2) ) / (2*sigma2) );
            kernel.at<float>(r+i,r+j)=value;
            kernel.at<float>(r-i,r+j)=value;
            kernel.at<float>(r+i,r-j)=value;
            kernel.at<float>(r-i,r-j)=value;
        }
    }

    float s = sum(kernel)[0];
    kernel/=s;

    return kernel;
}

//ex2
void imagesc(std::string name, cv::Mat mat){
    double min,max;
    cv::minMaxLoc(mat,&min,&max);
    cv::Mat  kernel_prime = mat/max;
    showImage(name, kernel_prime, 50,50);
}

//ex7
__global__ void convolutionGlobal(float *imgIn, float *GK, float *imgOut, int w, int h, int nc, int kernelSize){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;
    size_t k = kernelSize;

    int rx=k/2;
    int ry=k/2;

    if(x>=w || y>=h) return; //check for blocks

    for(unsigned int c=0;c<nc;c++) {
        float sum=0;
        for(unsigned int i=0;i<k;i++){
            unsigned int x_new;
            if(x+rx<i) x_new=rx;
            else if(x+rx-i>=w) x_new=w+rx-1;
            else x_new=x+rx-i;
            for(unsigned int j=0;j<k;j++){
                unsigned int y_new;
                if(y+ry<j) y_new=ry;
                else if(y+ry-j>=h) y_new=h+ry-1;
                else y_new=y+ry-j;
                sum+=GK[i+j*k]*imgIn[x_new+y_new*w+w*h*c];
            }
        }
        imgOut[x+w*y+w*h*c]=sum;
    }
}

__global__ void convolutionShared(float *imgIn, float *kernel, float *imgOut, Params params){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    extern __shared__ float shmem[];

    int r=params.r;
    int nc=params.nc;
    int w=params.w;
    int h=params.h;
    int shw=params.shw;
    int shh=params.shh;


    int tx=threadIdx.x;
    int bx=blockIdx.x;
    int ty=threadIdx.y;
    int by=blockIdx.y;   
    int bdx=blockDim.x;
    int bdy=blockDim.y;   

    for(unsigned int c=0;c<nc;c++,__syncthreads()) {

        ////////
        //step 1: copy data into shared memory, with clamping padding
        //
        for(int pt=tx+bdx*ty ; pt<shw*shh ;pt+=bdx*bdy){
            int xi = (pt % shw) + (bx *bdx - r);
            int yi = (pt / shw) + (by *bdy - r);

            xi = max(min(xi,w-1),0);
            yi = max(min(yi,h-1),0);

            float val=imgIn[xi + yi*w + c*w*h];

            shmem[pt] = val; 
        }

        __syncthreads();


        ///////
        //step 2: convolution, no more clamping needed
        //
        if(x>=w || y>=h) continue; //check for block border only AFTER copying to shared mem (goes over block borders)

        float sum=0;

        //convolution using adrian + markus indexing
	int kernelSize=2*r+1;
	for(int i=0;i<kernelSize;i++){
	  for(int j=0;j<kernelSize;j++){
	    int x_new=threadIdx.x+i;
	    int y_new=threadIdx.y+j;
	    sum+=kernel[i+j*kernelSize]*shmem[x_new+y_new*shw];
	  }
	}
        imgOut[x+w*y+w*h*c]=sum;
    }
}

__global__ void convolutionTexture(float *imgOut, float *kernel, int w, int h, int nc, int kernelSize) {
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;
    size_t k = kernelSize;

    int rx=kernelSize/2;
    int ry=kernelSize/2;

    if(x>=w || y>=h) return; //check for blocks

    for(size_t c=0;c<nc;c++) {
        float sum=0;
        for(size_t i=0;i<k;i++){
            size_t x_new;
	    x_new=x+rx-i;
	    
            for(size_t j=0;j<k;j++){
                size_t y_new;
		y_new=y+ry-j;

		float x_tex = x_new + 0.5f;
		float y_tex = y_new + c*h;

                sum+=kernel[i+j*k]*tex2D(texRef, x_tex, y_tex);
            }
        }
        imgOut[x+w*y+w*h*c]=sum;
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    float sigma=3.0f;
    getParam("sigma", sigma, argc, argv);
    if(sigma<0) sigma=3.0f;
    cout << "sigma: " << sigma << endl;


    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels

    // Set the output image format
    cv::Mat mShared(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mGlobal(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mTexture(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[n];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgShared = new float[n];
    float *imgGlobal = new float[n];
    float *imgTexture = new float[n];

    size_t n1 = (size_t)w*h*1;
    float *imgKernel  = new float[n1];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    int r = ceil(3.0f*sigma);
    cv::Mat k=kernel(sigma,r);

    // show input image
    showImage("Input image", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered(imgIn, mIn);
    convert_mat_to_layered(imgKernel,k);

    assert(k.rows == k.cols);

    float *d_imgIn, *d_imgKernel;
    float *d_imgShared, *d_imgGlobal, *d_imgTexture;
    Params *d_params;

    dim3 block = dim3(32,4,1); //32,16 for birds eye
    dim3 grid = dim3((w + block.x - 1 ) / block.x,(h + block.y - 1 ) / block.y, 1);

    Params params;
    params.r=r;
    params.shw = (block.x + 2*r);
    params.shh = (block.y + 2*r);
    params.w = w;
    params.h = h;
    params.nc = nc;

    size_t smBytes = params.shw * params.shh * sizeof(float);

    hipMalloc(&d_params, sizeof(Params) );CUDA_CHECK;
    hipMemcpy(d_params, &params, sizeof(Params), hipMemcpyHostToDevice);CUDA_CHECK;

    hipMalloc(&d_imgIn, n * sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    size_t kernelSize = (2*r+1);
    hipMalloc(&d_imgKernel,  kernelSize * kernelSize* sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgKernel, imgKernel, kernelSize * kernelSize * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;

    hipMalloc(&d_imgShared, n * sizeof(float) ); CUDA_CHECK;
    hipMalloc(&d_imgGlobal, n * sizeof(float) ); CUDA_CHECK;
    hipMalloc(&d_imgTexture, n * sizeof(float) ); CUDA_CHECK;

    // now set up the texture stuff
    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    hipBindTexture2D(NULL, &texRef, d_imgIn, &desc, w, nc * h, w * sizeof(d_imgIn[0]));
    CUDA_CHECK;

    // do convolution with shared memory
    convolutionShared<<<grid,block,smBytes>>> (d_imgIn, d_imgKernel, d_imgShared, params);CUDA_CHECK;
    hipDeviceSynchronize(); CUDA_CHECK;
    hipMemcpy(imgShared, d_imgShared, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    convolutionGlobal<<<grid,block>>>(d_imgIn, d_imgKernel, d_imgGlobal, w, h, nc, kernelSize); CUDA_CHECK;
    hipDeviceSynchronize(); CUDA_CHECK;
    hipMemcpy(imgGlobal, d_imgGlobal, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    convolutionTexture<<<grid,block>>>(d_imgTexture, d_imgKernel, w, h, nc, kernelSize); CUDA_CHECK;
    hipDeviceSynchronize(); CUDA_CHECK;
    hipMemcpy(imgTexture, d_imgTexture, n * sizeof(float), hipMemcpyDeviceToHost); CUDA_CHECK;

    // unbind texture
    hipUnbindTexture(texRef);CUDA_CHECK;

    hipFree(d_imgIn);CUDA_CHECK;
    hipFree(d_imgKernel);CUDA_CHECK;
    hipFree(d_params);CUDA_CHECK;
    hipFree(d_imgShared);CUDA_CHECK;
    hipFree(d_imgGlobal);CUDA_CHECK;
    hipFree(d_imgTexture);CUDA_CHECK;

    convert_layered_to_mat(mShared, imgShared);
    showImage("Convolution Shared Memory", mShared, 100+w+40, 100);

    convert_layered_to_mat(mGlobal, imgGlobal);
    showImage("Convolution Global Memory", mGlobal, 100+2*w+40, 100);

    convert_layered_to_mat(mTexture, imgTexture);
    showImage("Convolution Texture Memory", mTexture, 100+3*w+40, 100);

    // convert_layered_to_mat(mTexture, imgTexture);
    // showImage("Convolution Texture Memory", mTexture, 100+3*w+40, 100);

    //cv::Mat blurred=convolution(k,mIn);
    // show output image: first convert to interleaved opencv format from the layered raw array
    //showImage("Blurred", blurred, 100+w+40, 100);
    //std::cout<<"after showing blurred image"<<std::cout;
    
    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    //cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgKernel;

    delete[] imgShared;
    delete[] imgTexture;
    delete[] imgGlobal;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
