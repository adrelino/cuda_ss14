#include "hip/hip_runtime.h"
// ### Adrian's
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, , p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

typedef struct Params {
    int shw;
    int shh;
    int w;
    int h;
    int nc;
    int r;
} Params;


//ex1
cv::Mat kernel(float sigma, int r){
    float sigma2=powf(sigma,2);

    cv::Mat kernel(2*r+1,2*r+1,CV_32FC1);

    if(r==0){
        kernel.at<float>(0,0)=1;
        return kernel;
    }

    for (int i = 0; i <= r; ++i)
    {
        for (int j = 0; j <= r; ++j)
        {
            float value=1/(2*M_PI*sigma2) * expf( -( powf(i,2)+powf(j,2) ) / (2*sigma2) );
            kernel.at<float>(r+i,r+j)=value;
            kernel.at<float>(r-i,r+j)=value;
            kernel.at<float>(r+i,r-j)=value;
            kernel.at<float>(r-i,r-j)=value;
        }
    }

    float s = sum(kernel)[0];
    kernel/=s;

    return kernel;
}

//ex2
void imagesc(std::string name, cv::Mat mat){
    double min,max;
    cv::minMaxLoc(mat,&min,&max);
    cv::Mat  kernel_prime = mat/max;
    showImage(name, kernel_prime, 50,50);
}

//ex7
__global__ void convolutionShared(float *imgIn, float *kernel, float *imgOut, Params params){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    extern __shared__ float shmem[];

    int r=params.r;
    int nc=params.nc;
    int w=params.w;
    int h=params.h;
    int shw=params.shw;
    int shh=params.shh;


    int tx=threadIdx.x;
    int bx=blockIdx.x;
    int ty=threadIdx.y;
    int by=blockIdx.y;   
    int bdx=blockDim.x;
    int bdy=blockDim.y;   

    for(unsigned int c=0;c<nc;c++,__syncthreads()) {

        ////////
        //step 1: copy data into shared memory, with clamping padding
        //
        for(int pt=tx+bdx*ty ; pt<shw*shh ;pt+=bdx*bdy){
            int xi = (pt % shw) + (bx *bdx - r);
            int yi = (pt / shw) + (by *bdy - r);

            xi = max(min(xi,w-1),0);
            yi = max(min(yi,h-1),0);

            float val=imgIn[xi + yi*w + c*w*h];

            shmem[pt] = val; 
        }

        __syncthreads();


        ///////
        //step 2: convolution, no more clamping needed
        //
        if(x>=w || y>=h) continue; //check for block border only AFTER copying to shared mem (goes over block borders)

        float sum=0;

        //convolution using adrian + markus indexing
	int kernelSize=2*r+1;
	for(int i=0;i<kernelSize;i++){
	  for(int j=0;j<kernelSize;j++){
	    int x_new=threadIdx.x+i;
	    int y_new=threadIdx.y+j;
	    sum+=kernel[i+j*kernelSize]*shmem[x_new+y_new*shw];
	  }
	}
        imgOut[x+w*y+w*h*c]=sum;
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    float sigma=3.0f;
    getParam("sigma", sigma, argc, argv);
    if(sigma<0) sigma=3.0f;
    cout << "sigma: " << sigma << endl;


    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOut3(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[n];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[n];

    size_t n1 = (size_t)w*h*1;
    float *imgKernel  = new float[n1];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    int r = ceil(3.0f*sigma);
    cv::Mat k=kernel(sigma,r);
    
    imagesc("Kernel", k);


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered(imgIn, mIn);
    convert_mat_to_layered(imgKernel,k);

    
    assert(k.rows == k.cols);

    float *d_imgIn, *d_imgKernel, *d_imgOut, *d_imgShared;
    Params *d_params;

    dim3 block = dim3(16,16,1); //32,16 for birds eye
    dim3 grid = dim3((w + block.x - 1 ) / block.x,(h + block.y - 1 ) / block.y, 1);

    Params params;
    params.r=r;
    params.shw = (block.x + 2*r);
    params.shh = (block.y + 2*r);
    params.w = w;
    params.h = h;
    params.nc = nc;


    size_t smBytes = params.shw * params.shh * sizeof(float);

    size_t n3 = (size_t)params.shw*params.shh*nc;
    float *imgShared  = new float[n3];
    cv::Mat mOut2(params.shh,params.shw,mIn.type());  // mOut will have the same number of channels as the input image, nc layers


    hipMalloc(&d_params, sizeof(Params) );CUDA_CHECK;
    hipMemcpy(d_params, &params, sizeof(Params), hipMemcpyHostToDevice);CUDA_CHECK;
    hipMalloc(&d_imgIn, n * sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;
    hipMalloc(&d_imgKernel, n * sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgKernel, imgKernel, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;
    hipMalloc(&d_imgShared, n3 * sizeof(float) );CUDA_CHECK;
    hipMalloc(&d_imgOut, n * sizeof(float) ); CUDA_CHECK;

    //adrians indexing: ok
    convolutionShared <<<grid,block,smBytes>>> (d_imgIn, d_imgKernel, d_imgOut, params);CUDA_CHECK;
    hipMemcpy(imgOut, d_imgOut, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;
    hipMemcpy(imgShared, d_imgShared, n3 * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;

    hipFree(d_imgIn);CUDA_CHECK;
    hipFree(d_imgOut);CUDA_CHECK;
    hipFree(d_imgKernel);CUDA_CHECK;
    hipFree(d_params);CUDA_CHECK;
    hipFree(d_imgShared);CUDA_CHECK;

    convert_layered_to_mat(mOut, imgOut);
    showImage("Convolution Shared Memory", mOut, 100+w+40, 100);

    //cv::Mat blurred=convolution(k,mIn);
    // show output image: first convert to interleaved opencv format from the layered raw array
    //showImage("Blurred", blurred, 100+w+40, 100);
    //std::cout<<"after showing blurred image"<<std::cout;
    
    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    //cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    delete[] imgKernel;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
