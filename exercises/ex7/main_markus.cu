#include "hip/hip_runtime.h"
// ### Adrian's
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2014, September 8 - October 10
// ###
// ###
// ### Maria Klodt, Jan Stuehmer, Mohamed Souiai, Thomas Moellenhoff
// ###
// ###

// ###
// ###
// ### TODO: For every student of your group, please provide here:
// ###
// ### name, email, login username (for example p123)
// ### Dennis Mack, dennis.mack@tum.de, p060
// ### Adrian Haarbach, haarbach@in.tum.de, p077
// ### Markus Schlaffer, , p070


#include "aux.h"
#include <iostream>
#include <math.h>
//#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

float gaussian(float x, float y, float sigma){
    return expf(-(x*x+y*y)/(2*sigma*sigma))/2.0f/M_PI/sigma/sigma;
}

void createKernel(float sigma, float* GK, int k){
    int k2=k/2;

    float sum=0;
    for(int i=0;i<k;i++){
        for(int j=0;j<k;j++){
            float tmp=gaussian(i-k2,j-k2, sigma);
            sum+=tmp;
            GK[i+j*k]=tmp;
        }
    }
    for(int i=0;i<k;i++){
        for(int j=0;j<k;j++){
            GK[i+j*k]/=sum;
        }
    }
}

//                       in             out      out
__global__ void convolutionGPU(float *imgIn, float *imgKernel, float *imgOut, int w, int h, int nc, int k){
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;
    size_t x_block = threadIdx.x;
    size_t y_block = threadIdx.y;

    int r=k/2;

    //NOTE: BLOCK NICHT QUADRATISCH
    int shw=blockDim.x+2*r;
    int shh=blockDim.y+2*r;
    int stepx=shw/blockDim.x+1;
    int stepy=shh/blockDim.y+1;

    int2 topleft;
    topleft.x=blockIdx.x*blockDim.x-r;
    topleft.y=blockIdx.y*blockDim.y-r;

    extern __shared__ float s_data[];

    for(unsigned int c=0;c<nc;c++) {
        //fill shared memory
        for(unsigned int i=0;i<stepx;i++){
            int i_new=topleft.x+x_block*stepx+i;
            // if(i_new<topleft.x) continue;
            // else 
            if(i_new>=topleft.x+shw) continue;
            else if(i_new<0) i_new=0;
            else if(i_new>=w) i_new=w-1;

            for(unsigned int j=0;j<stepy;j++){
                int j_new=topleft.y+y_block*stepy+j;
                // if(j_new<topleft.y) continue;
                // else 
                if(j_new>=topleft.y+shh) continue;
                else if(j_new<0) j_new=0;
                else if(j_new>=h) j_new=h-1;

                s_data[x_block*stepx+i+(y_block*stepy+j)*shw]=imgIn[i_new + w * j_new + h*w*c];
            }
        }
        
        __syncthreads();
        
        if(x>=w || y>=h) continue; 

        //convolution
        float sum=0;

        for(unsigned int i=0;i<k;i++){
            int x_new=x_block+k-i;
            if(x_new>=shw) x_new=shw-1;
            for(unsigned int j=0;j<k;j++){
                int y_new=y_block+k-j;
                if(y_new>=shh) y_new=shh-1;
                sum+=imgKernel[i+j*k]*s_data[x_new+y_new*shw];
            }
        }
        imgOut[x+w*y+w*h*c]=sum;

        // alt: output=input
        // imgOut[x+w*y+w*h*c]=s_data[(x_block+r)+(y_block+r)*shw];

        __syncthreads();
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    float sigma=3.0f;
    getParam("sigma", sigma, argc, argv);
    if(sigma<=0) sigma=3.0f;
    cout << "sigma: " << sigma << endl;


    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << " nc="<<nc <<endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers



    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    size_t n = (size_t)w*h*nc;
    float *imgIn  = new float[n];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[n];

    // size_t n1 = (size_t)w*h*1;
    // float *imgKernel  = new float[n1];








    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // cv::Mat M=kernel(sigma);

    // imagesc("Kernel", M);
    size_t r=ceil(3*sigma);
    size_t k=2*r+1;
    size_t k2=k*k;
    float *GK=new float[k2];
    createKernel(sigma, GK, k);

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    std::cout<<"after showing input image"<<std::cout;




    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered(imgIn, mIn);
    // convert_mat_to_layered(imgKernel,M);

	//GPU:

    // int k=M.cols;
    // int hk=k.rows;

	float *d_imgIn, *d_imgOut, *d_imgKernel;
	hipMalloc(&d_imgIn, n * sizeof(float) );CUDA_CHECK;
	hipMemcpy(d_imgIn, imgIn, n * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;
    hipMalloc(&d_imgKernel, k*k * sizeof(float) );CUDA_CHECK;
    hipMemcpy(d_imgKernel, GK, k*k * sizeof(float), hipMemcpyHostToDevice);CUDA_CHECK;
    hipMalloc(&d_imgOut, n * sizeof(float) ); CUDA_CHECK;
    hipMemset(d_imgOut, 0, n * sizeof(float)); CUDA_CHECK;

	dim3 block = dim3(32,8,1);
	dim3 grid = dim3((w + block.x - 1 ) / block.x,(h + block.y - 1 ) / block.y, 1);

    cout <<"grids: "<< grid.x<< "x" <<grid.y<<endl;

	size_t smBytes = (block.x+r+r) * (block.y+r+r) * block.z * sizeof(float);

    convolutionGPU <<<grid,block,smBytes>>> (d_imgIn, d_imgKernel, d_imgOut, w, h, nc, k);CUDA_CHECK;
	hipDeviceSynchronize();CUDA_CHECK;


	hipMemcpy(imgOut, d_imgOut, n * sizeof(float), hipMemcpyDeviceToHost);CUDA_CHECK;

	hipFree(d_imgIn);CUDA_CHECK;
    hipFree(d_imgOut);CUDA_CHECK;
    hipFree(d_imgKernel);CUDA_CHECK;

    convert_layered_to_mat(mOut, imgOut);
    showImage("Convolution GPU", mOut, 100+2*w+40, 100);


    //cv::Mat blurred=convolution(k,mIn);
    // show output image: first convert to interleaved opencv format from the layered raw array
    //showImage("Blurred", blurred, 100+w+40, 100);
    //std::cout<<"after showing blurred image"<<std::cout;
    
    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    //cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



